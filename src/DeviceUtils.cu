#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <optional>

#include "DeviceUtils.hpp"

#include "tracing/Camera.hpp"
#include "tracing/Objects.hpp"
#include "tracing/SampleScene.hpp"

void cudaCheckLAstError(const char *file, int line, bool abort)
{
    const auto code = hipPeekAtLastError();
    
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort) exit(code);
    }
}


CudaRandomStates::CudaRandomStates(Size2i resolution)
    : size(resolution)
    , rand_states(nullptr)
{
    hipMalloc(&rand_states, resolution.width * resolution.height * sizeof(*rand_states));

    init();
}

CudaRandomStates::~CudaRandomStates()
{
    hipFree(rand_states);
    rand_states = nullptr;
}


void printCudaDeviceInfo() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return;
    }

    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    std::cout << "CUDA Device Info:" << std::endl;
    std::cout << "Name: " << deviceProp.name << std::endl;
    std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
}
