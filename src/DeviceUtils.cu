#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <optional>

#include "DeviceUtils.hpp"

#include "tracing/Camera.hpp"
#include "tracing/Objects.hpp"
#include "tracing/SampleScene.hpp"

void cudaCheckLAstError(const char *file, int line, bool abort)
{
    const auto code = hipPeekAtLastError();
    
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort) exit(code);
    }
}


CudaRandomStates::CudaRandomStates(Size2i resolution)
    : size(resolution)
    , rand_states(nullptr)
{
    hipMalloc(&rand_states, resolution.width * resolution.height * sizeof(*rand_states));

    init();
}

CudaRandomStates::~CudaRandomStates()
{
    hipFree(rand_states);
    rand_states = nullptr;
}


void printCudaDeviceInfo() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return;
    }

    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    std::cout << "CUDA Device Info:" << std::endl;
    std::cout << "Name: " << deviceProp.name << std::endl;
    std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
}

__global__ void initRand(hiprandState *randStates, int width, int height, unsigned long seed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    hiprand_init(seed, idx, 0, &randStates[idx]);
}

void CudaRandomStates::init()
{
    dim3 dimBlock(32, 32);
    dim3 dimGrid;
    dimGrid.x = (size.width + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (size.height + dimBlock.y - 1) / dimBlock.y;

    initRand<<<dimGrid, dimBlock>>>(rand_states, size.width, size.height, 42);
    CUDA_ERROR_CHECK();
}


#include "DeviceVectorImpl.hpp"
